#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16
#define GROUP_SIZE 2

__global__ void mandelKernel(int resX, int resY, float stepX, float stepY, int *GPUresult, float lowerX, float lowerY, int maxIterations, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * GROUP_SIZE;
    float x;
    float y;
    float z_re, z_im;

    for(int i = 0; i < GROUP_SIZE; i++){
        for(int j = 0; j < GROUP_SIZE; j++){
            if(thisX + i >= resX || thisY + j >= resY)
                continue;
            x = lowerX + (thisX+i) * stepX;
            y = lowerY + (thisY+j) * stepY;
            z_re = x, z_im = y;
            int t = 0;
            for (t = 0; t < maxIterations; ++t){
                if (z_re * z_re + z_im * z_im > 4.f)
                    break;
                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = x + new_re;
                z_im = y + new_im;
            }
            GPUresult[(thisY+j) * (pitch / sizeof(int)) + (thisX+i)] = t;
        }
    }    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    size_t pitch = 0;

    int size = resX * resY;
    int *result_h;
    int *result_d;
    hipHostAlloc((void **)&result_h, size * sizeof(int), hipHostMallocDefault);
    hipMallocPitch((void **)&result_d, &pitch, resX * sizeof(int), resY);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock((resX + GROUP_SIZE * BLOCK_SIZE - 1) / BLOCK_SIZE / GROUP_SIZE, (resY +  GROUP_SIZE * BLOCK_SIZE - 1) / BLOCK_SIZE / GROUP_SIZE);

    mandelKernel<<<numBlock, blockSize>>>(resX, resY, stepX, stepY, result_d, lowerX, lowerY, maxIterations, pitch);

    hipDeviceSynchronize();
    hipMemcpy2D(result_h, resX * sizeof(int), result_d, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    hipMemcpy(img, result_h, size * sizeof(int), hipMemcpyHostToHost);

    hipHostFree(result_h);
    hipFree(result_d);
}
