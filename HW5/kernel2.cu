#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16

__global__ void mandelKernel(int resX, int resY, float stepX, float stepY, int *GPUresult, float lowerX, float lowerY, int maxIterations, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    if(thisX < resX && thisY < resY){
        float x = lowerX + thisX * stepX;
        float y = lowerY + thisY * stepY;
        float z_re = x, z_im = y;
        int i;
        for (i = 0; i < maxIterations; ++i){
            if (z_re * z_re + z_im * z_im > 4.f)
                break;
            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = x + new_re;
            z_im = y + new_im;
        }
        *((int*)((char*)GPUresult + thisY * pitch) + thisX) = i;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    size_t pitch = 0;

    int size = resX * resY;
    int *result_h;
    int *result_d;
    hipHostAlloc((void **)&result_h, size * sizeof(int), hipHostMallocDefault);
    hipMallocPitch((void **)&result_d, &pitch, resX * sizeof(int), resY);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock((resX + BLOCK_SIZE - 1) / BLOCK_SIZE, (resY + BLOCK_SIZE - 1) / BLOCK_SIZE);

    mandelKernel<<<numBlock, blockSize>>>(resX, resY, stepX, stepY, result_d, lowerX, lowerY, maxIterations, pitch);

    hipDeviceSynchronize();
    hipMemcpy2D(result_h, resX * sizeof(int), result_d, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    hipMemcpy(img, result_h, size * sizeof(int), hipMemcpyHostToHost);

    hipHostFree(result_h);
    hipFree(result_d);
}
